#include "hip/hip_runtime.h"
#include "./hadamard_sparse_op.h"
#include "./elemwise_binary_op.h"
#include "./elemwise_binary_broadcast_op.h"
#include "../operator_common.h"
#include <mshadow/tensor.h>
#include "../mshadow_op.h"
#include "../elemwise_op_common.h"
#include "./elemwise_binary_op.h"

#include "../mxnet_op.h"
#include "broadcast_reduce_op.h"

#define WARPS_PER_BLOCK 1
#define THREADS_PER_BLOCK 256


namespace mshadow {
namespace cuda {




template <typename DType>
__global__ void hadamard_sparse_forward_kernel(const int nthreads, DType *out, DType *indices, DType *value, DType *key, int in_dim, int out_dim, DType *sign, DType *save) {


   const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= nthreads){
         return;
     }

    int k = out_dim;
    int col = index%out_dim;
    //int nnz = in_dim;

    DType *pKeys = key;
    DType *pIndices = indices;
    int sample = index/out_dim;
    int start;
    if (sample==0){
        start = 0;
        
    }else{
        start = *(save+sample-1); 
    }
    
    int end = *(save+sample);
    DType *pValues = value+start;

   
    for (int j = start; j<end; j++) {

            int ind = (int) *(pIndices+col);
            int row = (int) *(pKeys+j*2);
            int keyvalue = (int) *(pKeys+j*2+1);
            int signvalue = (int) *(sign+ keyvalue);
            DType *pRes = out;
            pRes += row*k+col;
            //printf("hello everyone %d %d %d %d %d\n", index, ind, j, row, keyvalue);
            *pRes += ((__popcll(ind & keyvalue) & 1) * -2 + 1) * (*pValues)* signvalue;

            //pKeys+=2;
            pValues++;

    }
}


template <typename DType>
inline void hadamardTransformGSparse(Tensor<gpu, 2, DType> &out, Tensor<gpu, 1, DType> &value, Tensor<gpu, 2, DType> &key, Tensor<gpu, 1, DType> &indices, Tensor<gpu, 1, DType> &sign, Tensor<gpu, 1, DType> &save) {

    int in_dim = (unsigned int) key.shape_[0];
    int n_samples = (unsigned int) out.shape_[0];
    int out_dim = (unsigned int) indices.shape_[1];
    out = 0;
    DType *out_p = out.dptr_;
    DType *value_p = value.dptr_;
    DType *key_p = key.dptr_;
    DType *sign_p = sign.dptr_;
    DType *save_p = save.dptr_;

    DType *indices_p = indices.dptr_;
    int processing_batch_size = 32;

    int upper_bound = n_samples/processing_batch_size;
    if (n_samples%processing_batch_size == 0){
      upper_bound = upper_bound-1;
    }
    upper_bound = upper_bound>0? upper_bound:0;

    int bstart = 0;
    //for ( int i = 0; i <= upper_bound; i++ ){
    //    int batchlen = min(processing_batch_size, n_samples - bstart );
        int nthreads = out_dim*n_samples;
        int threads_per_block = min(THREADS_PER_BLOCK, nthreads);
        int nblocks = (nthreads + threads_per_block - 1) / threads_per_block ;

        //printf("n_samples %d  upper_bound %d, nthreads %d, nblocks %d", n_samples, upper_bound, nthreads, nblocks);
        //LOG(INFO)<<out_dim<<in_dim<<nthreads<<threads_per_block<<nblocks;

        hadamard_sparse_forward_kernel<DType><<<nblocks, threads_per_block>>>(nthreads, out_p, indices_p, value_p, key_p, in_dim, out_dim, sign_p, save_p);
   //     bstart = (i+1)*batchlen;

   // }

}
}
}

namespace mxnet {
namespace op {


template<typename xpu>
void hadamardTransformGeneralSparse(const nnvm::NodeAttrs& attrs,
                       const OpContext& ctx,
                       const std::vector<TBlob>& inputs,
                       const std::vector<OpReqType>& req,
                       const std::vector<TBlob>& outputs) {
    using namespace mshadow;
    using namespace mshadow::expr;

    CHECK_EQ(inputs.size(), 5);
    CHECK_EQ(outputs.size(), 1);
    Stream<xpu> *s = ctx.get_stream<xpu>();

    MSHADOW_TYPE_SWITCH(outputs[0].type_flag_, DType, {

            Tensor<xpu, 2, DType> out = outputs[0].FlatTo2D<xpu, DType>(s);
            Tensor<xpu, 2, DType> key = inputs[0].FlatTo2D<xpu, DType>(s);
            Tensor<xpu, 1, DType> value = inputs[1].FlatTo1D<xpu, DType>(s);
            Tensor<xpu, 1, DType> indices = inputs[2].FlatTo1D<xpu, DType>(s);
            Tensor<xpu, 1, DType> sign = inputs[3].FlatTo1D<xpu, DType>(s);
            Tensor<xpu, 1, DType> workspace = inputs[4].FlatTo1D<xpu, DType>(s);
            // Tensor<xpu, 1, DType> workspace = ctx.requested[0].get_space_typed<xpu, 1, DType>(mshadow::Shape1(out.shape_[0]), s);
            mshadow::cuda::hadamardTransformGSparse<DType>(out, value, key,  indices, sign, workspace);


    });
}


NNVM_REGISTER_OP(hadamard_sparse)
.set_attr<FCompute>("FCompute<gpu>", hadamardTransformGeneralSparse<gpu>);

NNVM_REGISTER_OP(_backward_hadamard_sparse)
.set_attr<FCompute>("FCompute<gpu>", hadamardTransformGeneralSparse<gpu>);

}
}